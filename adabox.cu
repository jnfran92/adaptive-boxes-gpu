#include "hip/hip_runtime.h"

#include "stdio.h"
#include "stdlib.h"

//data
#include "./data/squares.h"



void get_right_bottom_rectangle(int idx_i_arg, int idx_j_arg, long m_arg, long n_arg, int *data_matrix_arg);
void get_left_bottom_rectangle(int idx_i_arg, int idx_j_arg, long m_arg, long n_arg, int *data_matrix_arg);
void get_left_top_rectangle(int idx_i_arg, int idx_j_arg, long n_arg, int *data_matrix_arg);
void get_right_top_rectangle(int idx_i_arg, int idx_j_arg, long n_arg, int *data_matrix_arg);

int main(){
	printf("adaptive-boxes-gpu\n");
	printf("GPU-accelerated rectangular decomposition for sound propagation modeling\n");

	printf("m %ld , n% ld\n",m, n);

	
	get_right_bottom_rectangle(0, 4, m, n, data);
	get_left_bottom_rectangle(0, 4, m, n, data);
	get_left_top_rectangle(0, 4, n, data);
	get_right_top_rectangle(0, 4, n, data);






	return 0;
}


void get_right_bottom_rectangle(int idx_i_arg, int idx_j_arg, long m_arg, long n_arg, int *data_matrix_arg){

	int step_j = 0;
	int first_step_i = 0;

	int i_val = 0;
	int j_val = 0;
	int temp_val = 0;
	int step_i = 0;
	
	int x1_val = 0;
	int x2_val = 0;
	int y1_val = 0;
	int y2_val = 0;


	while (true){
		i_val = idx_i_arg;
		j_val = idx_j_arg + step_j;

		if(j_val == n_arg){
			break;
		}

		temp_val = data_matrix_arg[i_val * n_arg + j_val];
		if (temp_val == 0){
			break;
		}

		step_i = 0;

		while (true){
			i_val = idx_i_arg + step_i;

			if (i_val == m_arg){
				break;
			}

			temp_val = data_matrix_arg[i_val * n_arg + j_val];

			if(temp_val == 0){
				break;
			}
			step_i++;
		}


		if (step_j == 0){
			first_step_i = step_i;
		}else{
			if(step_i < first_step_i){
				break;
			}
		}
		step_j++;
	}

	x1_val = idx_j_arg;
	y1_val = idx_i_arg;
	x2_val = idx_j_arg + step_j - 1;
	y2_val = idx_i_arg + first_step_i - 1;

	printf("x1 %d   x2 %d    y1 %d    y2 %d\n", x1_val, x2_val, y1_val, y2_val);

}


void get_left_bottom_rectangle(int idx_i_arg, int idx_j_arg, long m_arg, long n_arg, int *data_matrix_arg){

	int step_j = 0;
	int first_step_i = 0;

	int i_val = 0;
	int j_val = 0;
	int temp_val = 0;
	int step_i = 0;
	
	int x1_val = 0;
	int x2_val = 0;
	int y1_val = 0;
	int y2_val = 0;


	while (true){
		i_val = idx_i_arg;
		j_val = idx_j_arg - step_j;

		if(j_val == -1){
			break;
		}

		temp_val = data_matrix_arg[i_val * n_arg + j_val];
		if (temp_val == 0){
			break;
		}

		step_i = 0;
		while (true){
			i_val = idx_i_arg + step_i;

			if (i_val == m_arg){
				break;
			}

			temp_val = data_matrix_arg[i_val * n_arg + j_val];

			if(temp_val == 0){
				break;
			}
			step_i++;
		}


		if (step_j == 0){
			first_step_i = step_i;
		}else{
			if(step_i < first_step_i){
				break;
			}
		}
		step_j++;
	}

	x1_val = idx_j_arg;
	y1_val = idx_i_arg;
	x2_val = idx_j_arg - step_j + 1;
	y2_val = idx_i_arg + first_step_i - 1;

	printf("x1 %d   x2 %d    y1 %d    y2 %d\n", x1_val, x2_val, y1_val, y2_val);

}


void get_left_top_rectangle(int idx_i_arg, int idx_j_arg, long n_arg, int *data_matrix_arg){

	int step_j = 0;
	int first_step_i = 0;

	int i_val = 0;
	int j_val = 0;
	int temp_val = 0;
	int step_i = 0;
	
	int x1_val = 0;
	int x2_val = 0;
	int y1_val = 0;
	int y2_val = 0;


	while (true){
		i_val = idx_i_arg;
		j_val = idx_j_arg - step_j;

		if(j_val == -1){
			break;
		}

		temp_val = data_matrix_arg[i_val * n_arg + j_val];
		if (temp_val == 0){
			break;
		}

		step_i = 0;
		while (true){
			i_val = idx_i_arg - step_i;

			if (i_val == -1){
				break;
			}

			temp_val = data_matrix_arg[i_val * n_arg + j_val];

			if(temp_val == 0){
				break;
			}
			step_i++;
		}


		if (step_j == 0){
			first_step_i = step_i;
		}else{
			if(step_i < first_step_i){
				break;
			}
		}
		step_j++;
	}

	x1_val = idx_j_arg;
	y1_val = idx_i_arg;
	x2_val = idx_j_arg - step_j + 1;
	y2_val = idx_i_arg - first_step_i + 1;

	printf("x1 %d   x2 %d    y1 %d    y2 %d\n", x1_val, x2_val, y1_val, y2_val);

}


void get_right_top_rectangle(int idx_i_arg, int idx_j_arg, long n_arg, int *data_matrix_arg){

	int step_j = 0;
	int first_step_i = 0;

	int i_val = 0;
	int j_val = 0;
	int temp_val = 0;
	int step_i = 0;
	
	int x1_val = 0;
	int x2_val = 0;
	int y1_val = 0;
	int y2_val = 0;


	while (true){
		i_val = idx_i_arg;
		j_val = idx_j_arg + step_j;

		if(j_val == n_arg){
			break;
		}

		temp_val = data_matrix_arg[i_val * n_arg + j_val];
		if (temp_val == 0){
			break;
		}

		step_i = 0;

		while (true){
			i_val = idx_i_arg - step_i;

			if (i_val == -1){
				break;
			}

			temp_val = data_matrix_arg[i_val * n_arg + j_val];

			if(temp_val == 0){
				break;
			}
			step_i++;
		}


		if (step_j == 0){
			first_step_i = step_i;
		}else{
			if(step_i < first_step_i){
				break;
			}
		}
		step_j++;
	}

	x1_val = idx_j_arg;
	y1_val = idx_i_arg;
	x2_val = idx_j_arg + step_j - 1;
	y2_val = idx_i_arg - first_step_i + 1;

	printf("x1 %d   x2 %d    y1 %d    y2 %d\n", x1_val, x2_val, y1_val, y2_val);

}


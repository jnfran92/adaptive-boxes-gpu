#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "stdlib.h"
#include <iostream>
#include <fstream>
#include <string>
// thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/copy.h>
//STL
#include <vector>
// cuda call
#include "./include/cuda_call.h"
// kernels
#include "./include/rectangular_explorer_kernel.h"
#include "./include/rectangular_remover_kernel.h"
// rectangle struct
#include "./include/rectangle.h"
// data
#include "./data/theatre12.h"


int main(int argc, char *argv[]){
	printf("adaptive-boxes-gpu\n");
	printf("GPU-accelerated rectangular decomposition for sound propagation modeling\n");

	printf("----> Data size: m %ld , n% ld\n",m, n);

	// CUDA timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Rectangles vector
	std::vector<rectangle_t> recs;

	// CUDA
	//    number of tests = grid_x*grid_y	
	int grid_x = 4; // fixed
	int grid_y = atoi(argv[1]); //
	printf("----> Number of tests: %d \n",grid_x*grid_y);

	// GPU data
	int *data_d;
	int *areas_d;
	int *out_d;

	// Thrust Data
	thrust::device_vector<int> t_data_d(m*n);	
	data_d = thrust::raw_pointer_cast(&t_data_d[0]);

	thrust::device_vector<int> t_areas_d(grid_x*grid_y);
	areas_d = thrust::raw_pointer_cast(&t_areas_d[0]);

	thrust::device_vector<int> t_out_d(grid_x*grid_y*4);
	out_d = thrust::raw_pointer_cast(&t_out_d[0]);	
	
	// Get Mem
	/*hipMalloc((void **)&out_d, sizeof(int)*4*grid_x*grid_y);*/

	// CPU mem
	int *areas = new int[grid_x*grid_y];
	int *out = new int[4*grid_x*grid_y];

	// Copy data to device memory
	hipMemcpy(data_d, data, sizeof(int)*m*n, hipMemcpyHostToDevice);

	// Grid and Block size
	dim3 grid(grid_x, grid_y, 1);
	dim3 block(4, 1, 1); // fixed size
	
	dim3 image_grid(n/2,m/2,1);
	dim3 image_block(2,2,1);

	
	
	// Init algorithm -----------------------
	hipEventRecord(start);
	// Setup
	// hiprand
	hiprandState *devStates;
	CC(hipMalloc((void **)&devStates, grid_x*grid_y*sizeof(unsigned int)));
	
	setup_kernel<<<grid, block>>>(devStates);
	hipDeviceSynchronize();

	// Loop
	printf("Working...\n");
	rectangle_t rec;
	int max_step = 999999;
	int sum;
	// init last sum
	int last_sum = thrust::reduce(t_data_d.begin(), t_data_d.end());
	int last_x1 = -1;
	int last_x2 = -1;
	int last_y1 = -1;
	int last_y2 = -1;
	
	int x1,x2,y1,y2;

	for (int step=0; step<max_step; step++){
	printf("step\n");	
		find_largest_rectangle<<<grid,block>>>(devStates,m,n,data_d,out_d, areas_d);
		hipDeviceSynchronize();

	printf("step largest done \n");	
		thrust::device_vector<int>::iterator iter = thrust::max_element(t_areas_d.begin(), t_areas_d.end());
		unsigned int position = iter - t_areas_d.begin();
		int max_val = *iter; 
			
	printf("step max element done\n");	
		if (max_val==0){
			continue;
		}

		x1 = t_out_d[position*4 + 0];  
		x2 = t_out_d[position*4 + 1];  
		y1 = t_out_d[position*4 + 2];  
		y2 = t_out_d[position*4 + 3];  

	printf("get positions done\n");	

		if (!((last_x1==x1) & (last_x2==x2) & (last_y1==y1) & (last_y2==y2)) ){
			
			remove_rectangle_from_matrix<<<image_grid, image_block>>>(x1,x2,y1,y2, data_d, m, n);
			hipDeviceSynchronize();
			
			sum = thrust::reduce(t_data_d.begin(), t_data_d.end());
			hipDeviceSynchronize();
			printf("sum = %d\n",sum);			
			if(sum < last_sum){
				rec.x1 = x1;
				rec.x2 = x2;
				rec.y1 = y1;
				rec.y2 = y2;
				recs.push_back(rec);
			}
			
			last_sum = sum;
			if(sum<=0){
				break;
			}
			last_x1 = x1;
			last_x2 = x2;
			last_y1 = y1;
			last_y2 = y2;	
		}
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Decomposition ready!!\n");
	printf("-->Elapsed time: %f\n", milliseconds);
	printf("-->Last sum %d\n",sum);
	
	
	// Saving data in csv format
	std::ofstream r_file;
	std::string file_name = "./results/hall10_";
	file_name += std::to_string(grid_x*grid_y);
	file_name += ".csv";
	r_file.open(file_name);

	std::cout << "saving rectagles -  vector size "<< recs.size() << std::endl;
	std::vector<rectangle_t>::iterator v = recs.begin();
	while(v !=recs.end()){
		/*std::cout <<"  "<< v->x1 <<"  "<< v->x2 <<"  "<< v->y1 <<"  "<< v->y2 << std::endl;*/
		r_file << v->x1 <<",  "<< v->x2 <<",  "<< v->y1 <<",  "<< v->y2 << "\n";
		v++;
	}
	r_file.close();

	// free data
	delete areas;
	delete out;

	hipFree(devStates);

	return 0;
}
